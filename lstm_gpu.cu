#include "hip/hip_runtime.h"
#include "lstm_gpu.h"

#include <algorithm>
#include <cmath>
#include <cassert>
#include <cstdio>
#include <glog/logging.h>

#include "activation.h"
#include "cuda_utils.h"
#include "lstm_common.h"
#include "math_gpu.h"

#define MAX_STREAMS 1024

#define STREAMS_CREATE(N)                               \
  for (int z = 0; z < 4; ++z) {                         \
    for (int e = 0; e < (N); ++e) {                     \
      CHECK_CUDA_CALL(hipStreamCreate(&stream[z][e])); \
    }                                                   \
  }

#define STREAMS_DESTROY(N)                              \
  for (int z = 0; z < 4; ++z) {                         \
    for (int e = 0; e < (N); ++e) {                     \
      CHECK_CUDA_CALL(hipStreamDestroy(stream[z][e])); \
    }                                                   \
  }

#define STREAMS_SYNCHRONIZE(N)                                  \
  for (int z = 0; z < 4; ++z) {                                 \
    for (int e = 0; e < (N); ++e) {                             \
      CHECK_CUDA_CALL(hipStreamSynchronize(stream[z][e]));     \
    }                                                           \
  }

template <typename T>
__global__
void kernel_fill(const int n, T* x, const T v) {
  if (thGi >= n) return;
  x[thGi] = v;
}

template <typename T>
__global__
void kernel_init_Q_with_bias(
    const int H, const int W, const int N, const int K, const int D,
    const T* P, T* Q) {
  if (thGi >= 4 * H * W * N * 5 * D) return;
  const int d = thGi % D;                      // d \in [0 ... D-1]
  const int g = (thGi / D) % 5;                // g \in [0 ... 5]
  const int n = (thGi / (5 * D)) % N;          // n \in [0 ... N-1]
  const int x = (thGi / (N * 5 * D)) % W;      // x \in [0 ... W-1]
  const int y = (thGi / (W * N * 5 * D)) % H;  // y \in [0 ... H-1]
  const int z = (thGi / (H * W * N * 5 * D));  // z \in [0 ... 3]
  *Q_ptr(z, y, x, n, g, d) = *B_ptr(z, g, d);
}

template <typename T, typename FG, typename FI, typename FO>
__global__
void kernel_fw_elemwise_ops(const int H, const int W, const int N, const int D,
                            const int t, const int Tn, const int Tmin,
                            const int* S, T* Q, T* O) {
  if (thGi >= 4 * Tn * N * D) return;
  const int d = thGi % D;
  const int n = (thGi / D) % N;
  const int e = (thGi / (N * D)) % Tn;
  const int z = (thGi / (Tn * N * D));
  const int i = e + Tmin;
  const int j = t - i;
  const int y  = (z == 0 || z == 1) ? i : H - i - 1;
  const int x  = (z == 0 || z == 2) ? j : W - j - 1;
  const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;
  const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;
  if (S == nullptr || (y < S[n * 2] && x < S[n * 2 + 1])) {
    const T f_a   = FI::f(*Q_ptr(z, y, x, n, 0, d));  // f_i(input)
    const T f_gi  = FG::f(*Q_ptr(z, y, x, n, 1, d));  // f_g(input gate)
    const T f_go  = FG::f(*Q_ptr(z, y, x, n, 2, d));  // f_g(output gate)
    const T f_gfy = FG::f(*Q_ptr(z, y, x, n, 3, d));  // f_g(forget_y gate)
    const T f_gfx = FG::f(*Q_ptr(z, y, x, n, 4, d));  // f_g(forget_x gate)
    const T C_10  = (yp >= 0 && yp < H) ? *Q_ptr(z, yp, x, n, 5, d) : 0;
    const T C_01  = (xp >= 0 && xp < W) ? *Q_ptr(z, y, xp, n, 5, d) : 0;
    *Q_ptr(z, y, x, n, 5, d) = f_gi * f_a + f_gfy * C_10 + f_gfx * C_01;
    *O_ptr(y, x, n, z, d) = f_go * FO::f(*Q_ptr(z, y, x, n, 5, d));
  } else {
    *Q_ptr(z, y, x, n, 5, d) = 0;
    *O_ptr(y, x, n, z, d) = 0;
  }
}

template <typename T, typename FG, typename FI, typename FO>
__global__
void kernel_bw_elemwise_ops(const int H, const int W, const int N, const int D,
                            const int t, const int Tn, const int Tmin,
                            const int* S, const T* Q, T* dQ) {
  if (thGi >= 4 * Tn * N * D) return;
  const int d = thGi % D;
  const int n = (thGi / D) % N;
  const int e = (thGi / (N * D)) % Tn;
  const int z = (thGi / (Tn * N * D));
  const int i = e + Tmin;
  const int j = t - i;
  const int y = (z == 0 || z == 1) ? i : H - i - 1;
  const int x = (z == 0 || z == 2) ? j : W - j - 1;
  const int yn = (z == 0 || z == 1) ? y + 1 : y - 1;  // next y
  const int xn = (z == 0 || z == 2) ? x + 1 : x - 1;  // next x
  const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;  // previous y
  const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;  // previous x
  T* dA_00   = dQ_ptr(z, y, x, n, 0, d);
  T* dGi_00  = dQ_ptr(z, y, x, n, 1, d);
  T* dGo_00  = dQ_ptr(z, y, x, n, 2, d);
  T* dGfy_00 = dQ_ptr(z, y, x, n, 3, d);
  T* dGfx_00 = dQ_ptr(z, y, x, n, 4, d);
  T* dC_00   = dQ_ptr(z, y, x, n, 5, d);
  if (S == nullptr || (y < S[n * 2] && x < S[n * 2 + 1])) {
    const T dC_10 = (yn >= 0 && yn < H) ? *dQ_ptr(z, yn, x, n, 5, d) : 0;
    const T dC_01 = (xn >= 0 && xn < W) ? *dQ_ptr(z, y, xn, n, 5, d) : 0;
    const T Gfx_01 = (xn >= 0 && xn < W) ? *Q_ptr(z, y, xn, n, 4, d) : 0;
    const T Gfy_10 = (yn >= 0 && yn < H) ? *Q_ptr(z, yn, x, n, 3, d) : 0;
    const T C_10   = (yp >= 0 && yp < H) ? *Q_ptr(z, yp, x, n, 5, d) : 0;
    const T C_01   = (xp >= 0 && xp < W) ? *Q_ptr(z, y, xp, n, 5, d) : 0;
    const T C_00   = *Q_ptr(z, y, x, n, 5, d);
    const T Gfx_00 = *Q_ptr(z, y, x, n, 4, d);
    const T Gfy_00 = *Q_ptr(z, y, x, n, 3, d);
    const T Go_00  = *Q_ptr(z, y, x, n, 2, d);
    const T Gi_00  = *Q_ptr(z, y, x, n, 1, d);
    const T A_00   = *Q_ptr(z, y, x, n, 0, d);
    *dGo_00 = (*dC_00) * FO::f(C_00) * FG::df(Go_00);
    *dC_00  = (*dC_00) * FO::df(C_00) * FG::f(Go_00) +
        dC_10 * FG::f(Gfy_10) + dC_01 * FG::f(Gfx_01);
    *dGfy_00 =
        (yp >= 0 && yp < H) ? (*dC_00) * C_10 * FG::df(Gfy_00) : 0;
    *dGfx_00 =
        (xp >= 0 && xp < W) ? (*dC_00) * C_01 * FG::df(Gfx_00) : 0;
    *dGi_00  = (*dC_00) * FI::f(A_00) * FG::df(Gi_00);
    *dA_00   = (*dC_00) * FI::df(A_00) * FG::f(Gi_00);
  } else {
    *dA_00   = 0;
    *dGi_00  = 0;
    *dGo_00  = 0;
    *dGfy_00 = 0;
    *dGfx_00 = 0;
    *dC_00   = 0;
  }
}

template <typename T>
__global__
void kernel_copy_dO_to_dC(const int H, const int W, const int N, const int D,
                          const int t, const int Tn, const int Tmin,
                          const T* dO, T* dQ) {
  if (thGi >= 4 * Tn * N * D) return;
  const int d = thGi % D;
  const int n = (thGi / D) % N;
  const int e = (thGi / (N * D)) % Tn;
  const int z = (thGi / (Tn * N * D));
  const int i = e + Tmin;
  const int j = t - i;
  const int y = (z == 0 || z == 1) ? i : H - i - 1;
  const int x = (z == 0 || z == 2) ? j : W - j - 1;
  *dQ_ptr(z, y, x, n, 5, d) = *dO_ptr(y, x, n, z, d);
}


/* 2D-LSTM forward pass running on the GPU
 * H -> maximum height
 * W -> maximum width
 * N -> batch size
 * K -> input dimensions/channels
 * D -> output dimensions/channels
 * I -> input data (layout: H x W x N x K)
 * S -> input sizes (height and width of each sample, layout: N x 2)
 * P -> parameters (size: 4 * (1 + K + D + D) * 5 * D)
 * O -> output data (layout: H x W x N x 4 x D)
 * Q -> gates pre-activations and cells (layout: 4 x H x W x N x 6 x D)
 */
template <typename T, typename FG, typename FI, typename FO>
inline void fw_training(
    const int H, const int W, const int N, const int K, const int D,
    const T* I, const int* S, const T* P, T* O, T* Q) {
  CHECK_NOTNULL(I);
  CHECK_NOTNULL(P);
  CHECK_NOTNULL(O);
  CHECK_NOTNULL(Q);
  const int NSZ = std::max(std::min(std::min(H, W), MAX_STREAMS), 4);
  // Prepare cublas handler and streams
  hipblasHandle_t handle;
  CHECK_CUBLAS_CALL(hipblasCreate(&handle));
  hipStream_t stream[4][MAX_STREAMS];
  STREAMS_CREATE(NSZ);

  // Initialize gates with bias
  kernel_init_Q_with_bias<T><<<DIV_UP(4 * H * W * N * 5 * D, 512), 512>>>(
      H, W, N, K, D, P, Q);
  CHECK_LAST_CUDA_CALL();

  // Multiply inputs by weights.
  for (int z = 0; z < 4; ++z) {
    hipblasSetStream(handle, stream[z][0]);
    CHECK_CUBLAS_CALL(
        gemm_gpu<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, H * W * N, 5 * D, K,
                    1.0, I, K,
                    W_ptr(z, 0, 0, 0), 5 * D,
                    1.0, Q_ptr(z, 0, 0, 0, 0, 0), 6 * D));
  }

  // Synchronize streams
  STREAMS_SYNCHRONIZE(1);

  // Process the image diagonal-wise (there are H + W - 1 diagonals to process)
  for (int t = 0; t < H + W - 1; ++t) {
    // Compute number of elements in the u-th diagonal
    const int Tmin = std::max(0, t - W + 1);
    const int Tmax = std::min(t, H - 1);
    const int Tn   = (Tmax - Tmin) + 1;

    for (int z = 0; z < 4; ++z) {
      for (int e = 0; e < Tn; ++e) {
        CHECK_CUBLAS_CALL(hipblasSetStream(handle, stream[z][e % NSZ]));
        // (y, x) coordinates of the e-th element in the z-th diagonal.
        const int i = e + Tmin;
        const int j = t - i;
        const int y  = (z == 0 || z == 1) ? i : H - i - 1;
        const int x  = (z == 0 || z == 2) ? j : W - j - 1;
        const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;
        const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;
        T* Q_00 = Q_ptr(z, y, x, 0, 0, 0);
        if (yp >= 0 && yp <= H - 1) {
          const T* O_10 = O_ptr(yp, x, 0, z, 0);
          CHECK_CUBLAS_CALL(
              gemm_gpu<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 5 * D, D,
                          1.0, O_10, 4 * D,
                          Ry_ptr(z, 0, 0, 0), 5 * D,
                          1.0, Q_00, 6 * D));
        }
        if (xp >= 0 && xp <= W - 1) {
          const T* O_01 = O_ptr(y, xp, 0, z, 0);
          CHECK_CUBLAS_CALL(
              gemm_gpu<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 5 * D, D,
                          1.0, O_01, 4 * D,
                          Rx_ptr(z, 0, 0, 0), 5 * D,
                          1.0, Q_00, 6 * D));
        }
      }
    }
    STREAMS_SYNCHRONIZE(Tn);
    kernel_fw_elemwise_ops<T, FG, FI, FO><<<DIV_UP(4 * Tn * N * D, 512), 512>>>(
        H, W, N, D, t, Tn, Tmin, S, Q, O);
  }

  STREAMS_DESTROY(NSZ);
  CHECK_CUBLAS_CALL(hipblasDestroy(handle));  // TODO: check for errors
}


/* 2D-LSTM backward pass running on the GPU
 * H -> maximum height
 * W -> maximum width
 * N -> batch size
 * K -> input dimensions/channels
 * D -> output dimensions/channels
 * I -> input data (layout: H x W x N x K)
 * S -> input sizes (height and width of each sample, layout: N x 2)
 * P -> parameters (size: 4 * (1 + K + D + D) * 5 * D)
 * O -> output data (layout: H x W x N x 4 x D)
 * Q -> gates pre-activations and cells (layout: 4 x H x W x N x 6 x D)
 * dO -> derivative of the loss w.r.t the output
 * dQ -> derivative of the loss w.r.t the internal states
 */
template <typename T, typename FG, typename FI, typename FO>
inline void bw_workspace(
    const int H, const int W, const int N, const int K, const int D,
    const T* I, const int* S, const T* P, const T* O, const T* Q, const T* dO,
    T* dQ) {
  CHECK_NOTNULL(I);
  CHECK_NOTNULL(P);
  CHECK_NOTNULL(O);
  CHECK_NOTNULL(Q);
  CHECK_NOTNULL(dO);
  CHECK_NOTNULL(dQ);
  const int NSZ = std::max(std::min(std::min(H, W), MAX_STREAMS), 4);
  // Prepare cublas handler and streams
  hipblasHandle_t handle;
  CHECK_CUBLAS_CALL(hipblasCreate(&handle));
  hipStream_t stream[4][MAX_STREAMS];
  STREAMS_CREATE(NSZ);

  // Process the image diagonal-wise, in backwards order (there are H + W - 1
  // diagonals to process)
  for (int t = H + W - 2; t >= 0; --t) {
    // Compute number of elements in the diagonal
    const int Tmin = std::max(0, t - W + 1);
    const int Tmax = std::min(t, H - 1);
    const int Tn   = (Tmax - Tmin) + 1;
    kernel_copy_dO_to_dC<T><<<DIV_UP(4 * Tn * N * D, 512), 512>>>(
        H, W, N, D, t, Tn, Tmin, dO, dQ);
    CHECK_LAST_CUDA_CALL();

    for (int z = 0; z < 4; ++z) {
      for (int e = 0; e < Tn; ++e) {
        CHECK_CUBLAS_CALL(hipblasSetStream(handle, stream[z][e % NSZ]));
        const int i = e + Tmin;
        const int j = t - i;
        const int y  = (z == 0 || z == 1) ? i : H - i - 1;
        const int x  = (z == 0 || z == 2) ? j : W - j - 1;
        const int yn = (z == 0 || z == 1) ? y + 1 : y - 1;  // next y
        const int xn = (z == 0 || z == 2) ? x + 1 : x - 1;  // next x
        if (yn >= 0 && yn < H) {
          CHECK_CUBLAS_CALL(
              gemm_gpu<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, D, 5 * D,
                          1.0, dQ_ptr(z, yn, x, 0, 0, 0), 6 * D,
                          Ry_ptr(z, 0, 0, 0), 5 * D,
                          1.0, dQ_ptr(z, y, x, 0, 5, 0), 6 * D));
        }
        if (xn >= 0 && xn < W) {
          CHECK_CUBLAS_CALL(
              gemm_gpu<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, D, 5 * D,
                          1.0, dQ_ptr(z, y, xn, 0, 0, 0), 6 * D,
                          Rx_ptr(z, 0, 0, 0), 5 * D,
                          1.0, dQ_ptr(z, y, x, 0, 5, 0), 6 * D));
        }
      }
    }
    STREAMS_SYNCHRONIZE(Tn);
    kernel_bw_elemwise_ops< T, FG, FI, FO >
        <<<DIV_UP(4 * Tn * N * D, 512), 512>>>(
            H, W, N, D, t, Tn, Tmin, S, Q, dQ);
    CHECK_LAST_CUDA_CALL();
  }

  STREAMS_DESTROY(NSZ);
  CHECK_CUBLAS_CALL(hipblasDestroy(handle));
}

template <typename T>
inline void bw_input(
    const int H, const int W, const int N, const int K, const int D,
    const T* P, const T* dQ, const T scale, T* dI) {
  CHECK_NOTNULL(P);
  CHECK_NOTNULL(dQ);
  CHECK_NOTNULL(dI);
  // dJ/dI(y,x)
  hipblasHandle_t handle;
  hipblasCreate(&handle);  // TODO: check for errors
  for (int z = 0; z < 4; ++z) {
    gemm_gpu<T>(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, H * W * N, K, 5 * D,
                scale, dQ_ptr(z, 0, 0, 0, 0, 0), 6 * D,
                W_ptr(z, 0, 0, 0), 5 * D,
                1.0, dI, K);
  }
  hipblasDestroy(handle);  // TODO: check for errors
}

template <typename T>
inline void bw_param(
    const int H, const int W, const int N, const int K, const int D,
    const T* I, const T* O, const T* dQ, const T scale, T* dP) {
  CHECK_NOTNULL(I);
  CHECK_NOTNULL(O);
  CHECK_NOTNULL(dQ);
  CHECK_NOTNULL(dP);

  hipblasHandle_t handle;
  hipblasCreate(&handle);  // TODO: check for errors
  hipStream_t stream[4][4];
  STREAMS_CREATE(4);

  // dJ/db
  T* vOnes = nullptr;
  hipMalloc(&vOnes, sizeof(T) * H * W * N);
  kernel_fill<T><<<DIV_UP(H * W * N, 512), 512>>>(H * W * N, vOnes, 1);
  CHECK_LAST_CUDA_CALL();
  for (int z = 0; z < 4; ++z) {
    hipblasSetStream(handle, stream[z][0]);
    gemv_gpu<T>(handle, HIPBLAS_OP_T, H * W * N, 5 * D,
                scale, dQ_ptr(z, 0, 0, 0, 0, 0), 6 * D,
                vOnes, 1,
                1.0, dB_ptr(z, 0, 0), 1);
  }

  // dJ/dW
  for (int z = 0; z < 4; ++z) {
    hipblasSetStream(handle, stream[z][1]);
    gemm_gpu<T>(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, K, 5 * D, H * W * N,
                scale, I, K,
                dQ_ptr(z, 0, 0, 0, 0, 0), 6 * D,
                1.0, dW_ptr(z, 0, 0, 0), 5 * D);
  }

  // dJ/dRy
  for (int z = 0; z < 4; ++z) {
    hipblasSetStream(handle, stream[z][2]);
    for (int y = 0; y < H; ++y) {
      for (int x = 0; x < W; ++x) {
        const int yp = (z == 0 || z == 1) ? y - 1 : y + 1;  // previous y
        if (yp >= 0 && yp < H) {
          gemm_gpu<T>(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, D, 5 * D, N,
                      scale, O_ptr(yp, x, 0, z, 0), 4 * D,
                      dQ_ptr(z, y, x, 0, 0, 0), 6 * D,
                      1.0, dRy_ptr(z, 0, 0, 0), 5 * D);
        }
      }
    }
  }

  // dJ/dRx
  for (int z = 0; z < 4; ++z) {
    hipblasSetStream(handle, stream[z][3]);
    for (int y = 0; y < H; ++y) {
      for (int x = 0; x < W; ++x) {
        const int xp = (z == 0 || z == 2) ? x - 1 : x + 1;  // previous x
        if (xp >= 0 && xp < W) {
          gemm_gpu<T>(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, D, 5 * D, N,
                      scale, O_ptr(y, xp, 0, z, 0), 4 * D,
                      dQ_ptr(z, y, x, 0, 0, 0), 6 * D,
                      1.0, dRx_ptr(z, 0, 0, 0), 5 * D);
        }
      }
    }
  }

  STREAMS_SYNCHRONIZE(4);
  STREAMS_DESTROY(4);
  hipblasDestroy(handle);  // TODO: check for errors
}

extern "C" {
  DEFINE_WRAPPERS(gpu, float)
  DEFINE_WRAPPERS(gpu, double)
}  // extern "C"
